#include "hip/hip_runtime.h"
#include <stdio.h>

/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

__global__
void minimum(float *d_min_logLum, const float* const d_logLuminance){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int tidx = threadIdx.x;

    __shared__
        float smem[1024];

    smem[tidx] = d_logLuminance[tid];
    __syncthreads();

    int s = blockDim.x / 2;

    while(s >= 1){
        if(tidx < s){
            smem[tidx] = min(smem[tidx] , smem[tidx + s]);
        }
        __syncthreads();
        s >>= 1;
    }

    if(tidx == 0){
        d_min_logLum[blockIdx.x] = smem[0];
    }
}


__global__
void maximum(float *d_max_logLum, const float* const d_logLuminance){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int tidx = threadIdx.x;

    __shared__
        float smem[1024];

    smem[tidx] = d_logLuminance[tid];
    __syncthreads();

    int s = blockDim.x / 2;

    while(s >= 1){
        if(tidx < s){
            smem[tidx] = max(smem[tidx], smem[tidx + s]);
        }
        __syncthreads();
        s >>= 1;
    }

    if(tidx == 0){
        d_max_logLum[blockIdx.x] = smem[0];
    }
}
__global__
void hist(unsigned int* const d_hist, const float* const d_logLuminance, const float min_logLum, const float range, const int numBins, const int size){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < size){
        int bin = min(numBins - 1.0, (numBins * (d_logLuminance[tid] - min_logLum) / range));

        atomicAdd(&(d_hist[bin]), 1);
    }
}

__global__
void hillis_steele(unsigned int* const d_cdf, const unsigned int* const d_hist, int n){
    int tidx = threadIdx.x;

    if(tidx < n){
        __shared__
            unsigned int smem[2 * 1024];

        int head_in = 1;
        int head_out = 0;

        // right shift input, and store it in shared memory
        if(tidx == 0){
            smem[tidx] = 0;
        }
        else{
            smem[head_out * n + tidx] = d_hist[tidx - 1];
        }

        int offset = 1;

        while(offset < n){
            // swap buffer indices
            head_in = 1 - head_in;
            head_out = 1 - head_in;

            if(tidx < offset){
                smem[head_out * n + tidx] = smem[head_in * n + tidx];
            }else{
                smem[head_out * n + tidx] = smem[head_in * n + tidx] + smem[head_in * n + tidx - offset];
            }
            __syncthreads();

            offset <<= 1;
        }

        d_cdf[tidx] = smem[head_out * n + tidx];
    }
}

//uses 1024 threads
__global__
void blelloch1(unsigned int* const d_cdf, const unsigned int* const d_hist, int n){
    int tidx = threadIdx.x;

    if(tidx < n){
        int offset = 1;

        __shared__
            int smem[1024];

        smem[tidx] =  d_hist[tidx];
        __syncthreads();

        while(offset < n){
            bool condition = ((tidx + 1) % (2 * offset)) == 0;
            
            if(condition){
                smem[tidx] += smem[tidx - offset];
            }    
            __syncthreads();

            offset <<= 1;
        }

        if(tidx == n - 1){
            smem[tidx] = 0;
        }
        __syncthreads();

        offset = n >> 1;

        while(offset >= 1){
            bool condition = ((tidx + 1) % (2 * offset)) == 0;
            if(condition){
                int temp = smem[tidx - offset];
                smem[tidx - offset] = smem[tidx];
                smem[tidx] += temp;
            }
            __syncthreads();
            offset >>= 1;
        }

        d_cdf[tidx] = smem[tidx];
    }
}

// uses 512 threads
__global__
void blelloch2(unsigned int* const d_cdf, const unsigned int* const d_hist, int n){
    int tidx = threadIdx.x;
    int idx = 2 * tidx + 1;

    if(idx < n){
        int offset = 1;

        __shared__
            int smem[1024];

        smem[idx]     =  d_hist[idx];
        smem[idx - 1] =  d_hist[idx - 1];
        __syncthreads();

        while(offset < n){
            bool condition = ((idx + 1) % (2 * offset)) == 0;

            if(condition){
                smem[idx] += smem[idx - offset];
            }    
            __syncthreads();

            offset <<= 1;
        }

        if(idx == n - 1){
            smem[idx] = 0;
        }
        __syncthreads();

        offset = n >> 1;

        while(offset >= 1){
            bool condition = ((idx + 1) % (2 * offset)) == 0;
            if(condition){
                int temp = smem[idx - offset];
                smem[idx - offset] = smem[idx];
                smem[idx] += temp;
            }
            __syncthreads();
            offset >>= 1;
        }

        d_cdf[idx]     = smem[idx];
        d_cdf[idx - 1] = smem[idx - 1];
    }
}

// Parallel Prefix sum: Mark Harris
// uses 512 threads
__global__
void blelloch3(unsigned int* const d_cdf, const unsigned int* const d_hist, const int n){
  int tidx = threadIdx.x;
    
  if(tidx < n/2){
    __shared__
    int smem[1024];
     
    smem[ 2 * tidx ]     =  d_hist[ 2 * tidx ];
    smem[ 2 * tidx + 1 ] =  d_hist[ 2 * tidx + 1 ];
    __syncthreads();

    int d = n >> 1;
    int offset = 1;
    
    while(d > 0){
        if(tidx < d){
            int idx = offset * (2 * tidx + 2) - 1;
            smem[idx] += smem[idx - offset];
        }
        __syncthreads();

        offset <<= 1;
        d >>= 1;
    }

    if(tidx == 0){
      smem[n - 1] = 0;
    }
    __syncthreads();

    offset = n >> 1;
    d = 1;

    while(d < n){
        if(tidx < d){
            int idx = offset * (2 * tidx + 2) - 1;
            int temp = smem[idx - offset];
            smem[idx - offset] = smem[idx];
            smem[idx] += temp;
        }
        __syncthreads();
        offset >>= 1;
        d <<= 1;
    }

    d_cdf[2 * tidx]     = smem[2 * tidx];
    d_cdf[2 * tidx + 1] = smem[2 * tidx + 1];     
  }
}

void find_min(float *d_min_logLum, float *d_intermediate, const float* const d_logLuminance , const size_t size){
  const int maxThreadsPerBlock = 1024;

  int threads = maxThreadsPerBlock;
  int blocks = size / maxThreadsPerBlock;

  minimum<<<blocks, threads>>>(d_intermediate, d_logLuminance);

  blocks = 1;

  minimum<<<blocks, threads>>>(d_min_logLum, d_intermediate);
}

void find_max(float *d_max_logLum, float *d_intermediate, const float* const d_logLuminance, const size_t size){
  const int maxThreadsPerBlock = 1024;
  
  int threads = maxThreadsPerBlock;
  int blocks = size / maxThreadsPerBlock;

  maximum<<<blocks, threads>>>(d_intermediate, d_logLuminance);

  blocks = 1;

  maximum<<<blocks, threads>>>(d_max_logLum, d_intermediate);
}

void find_hist(unsigned int* const d_hist, const float* const d_logLuminance, const float min_logLum, const float range, const int numBins, const int size){
  int threads = 1024;
  int blocks = size/threads;

  hist<<<blocks, threads>>>(d_hist, d_logLuminance, min_logLum, range, numBins, size);
}

void prefix_sum(unsigned int* const d_cdf, const unsigned int* const d_hist, const int numBins){
  int threads = numBins;
  int blocks = 1;
  
  //hillis_steele<<<blocks, threads>>>(d_cdf, d_hist, numBins);
  //blelloch1<<<blocks, threads/2>>>(d_cdf, d_hist, numBins);
  blelloch2<<<blocks, threads/2>>>(d_cdf, d_hist, numBins);
  //blelloch3<<<blocks, threads/2>>>(d_cdf, d_hist, numBins);
}


#include "utils.h"

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  const size_t size = numRows * numCols;

  float *d_min_logLum;
  float *d_max_logLum;
  
  float range;

  float* d_intermediate;

  unsigned int *d_hist;

  unsigned int *h_hist = (unsigned int *)malloc(sizeof(unsigned int) * numBins);

  checkCudaErrors(hipMalloc(&d_min_logLum, sizeof(float)));
  checkCudaErrors(hipMalloc(&d_max_logLum,  sizeof(float)));
  
  checkCudaErrors(hipMalloc(&d_intermediate,  size * sizeof(float)));
  
  checkCudaErrors(hipMalloc(&d_hist, sizeof(unsigned int) * numBins));
  checkCudaErrors(hipMemset(d_hist, 0, sizeof(unsigned int) * numBins));
  
  find_min(d_min_logLum, d_intermediate, d_logLuminance, size);
  find_max(d_max_logLum, d_intermediate, d_logLuminance, size);
  
  checkCudaErrors(hipMemcpy(&min_logLum, d_min_logLum, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_max_logLum, sizeof(float), hipMemcpyDeviceToHost));

  range = max_logLum - min_logLum;

  find_hist(d_hist, d_logLuminance, min_logLum, range, numBins, size);
  checkCudaErrors(hipMemcpy(h_hist, d_hist, sizeof(unsigned int) * numBins, hipMemcpyDeviceToHost));

  prefix_sum(d_cdf, d_hist, numBins);
  
  /* Check cpu results: hist, cdf
  int sum = 0;
  for(int i = 0; i < 1024; ++i){
    sum += h_hist[i];
    printf("%d ", h_hist[i]);
  }
  printf("\nsize: %d, sum: %d\n", size, sum);

  int cdf_cpu[1024] = {0};

  int acc = 0;
  for(int i = 0; i < 1024; ++i){
    printf("%d ", acc);
    cdf_cpu[i] = acc;
    acc += h_hist[i];
  }
  printf("\n");
  */
  
  checkCudaErrors(hipFree(d_min_logLum));
  checkCudaErrors(hipFree(d_max_logLum));
}
